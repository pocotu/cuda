#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
using namespace std;
//-- Funcion device recursiva
__device__ int Factorial(int x) {
	if (x == 0)
		return 1;
	else
		return x * Factorial(x - 1);

}
//--kernel
__global__ void MyKernel(int n, int* fact) {
	*fact = Factorial(n);
}
int main() {
	int n, resultado, * d_f;
	//--Leer un entero
	//cout << "Ingrese un número: ";
	cin >> n;
	//--reservar memoria en el device
	hipMalloc(&d_f, sizeof(int));
	//--Lanzar el kernel
	MyKernel << <1, 1 >> > (n, d_f); //--1 bloque y un hilo por bloque
	//--Copiar el resultado al host
	hipMemcpy(&resultado, d_f, sizeof(int), hipMemcpyDeviceToHost);
	// Print out
	cout << "El resultado es: " << resultado << endl;
	hipFree(d_f); //--libera la memoria del host
	system("pause");
	return 0;

}